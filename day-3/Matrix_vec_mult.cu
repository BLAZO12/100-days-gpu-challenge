#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorMatrixMult(const float* matrix, const float* vector, float* result, int size) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < size) {
        float sum = 0.0f;
        for (int col = 0; col < size; col++) {
            sum += matrix[row * size + col] * vector[col];
        }
        result[row] = sum;
    }
}

int main() {
    const int size = 10;
    float *hostMatrix, *hostVector, *hostResult;

    std::cout << "Allocating host memory...\n";
    hostMatrix = (float *)malloc(size * size * sizeof(float));
    hostVector = (float *)malloc(size * sizeof(float));
    hostResult = (float *)malloc(size * sizeof(float));

    std::cout << "Initializing host data...\n";
    for (int i = 0; i < size; i++) {
        hostVector[i] = 2.0f;
        hostResult[i] = 0.0f;
        for (int j = 0; j < size; j++) {
            hostMatrix[i * size + j] = 1.0f;
        }
    }

    float *devMatrix, *devVector, *devResult;

    std::cout << "Allocating device memory...\n";
    hipMalloc(&devMatrix, size * size * sizeof(float));
    hipMalloc(&devVector, size * sizeof(float));
    hipMalloc(&devResult, size * sizeof(float));

    std::cout << "Copying data to device...\n";
    hipMemcpy(devMatrix, hostMatrix, size * size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(devVector, hostVector, size * sizeof(float), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (size + blockSize - 1) / blockSize;

    std::cout << "Launching kernel...\n";
    vectorMatrixMult<<<gridSize, blockSize>>>(devMatrix, devVector, devResult, size);
    hipDeviceSynchronize();

    std::cout << "Copying result back to host...\n";
    hipMemcpy(hostResult, devResult, size * sizeof(float), hipMemcpyDeviceToHost);

    std::cout << "\nMatrix A:\n";
    for (int i = 0; i < size; i++) {
        for (int j = 0; j < size; j++) {
            printf("%.2f ", hostMatrix[i * size + j]);
        }
        printf("\n");
    }

    std::cout << "\nVector B:\n";
    for (int i = 0; i < size; i++) {
        printf("%.2f ", hostVector[i]);
    }
    printf("\n");

    std::cout << "\nResult Vector C (Matrix x Vector):\n";
    for (int i = 0; i < size; i++) {
        printf("%.2f ", hostResult[i]);
    }
    printf("\n");

    std::cout << "Freeing device and host memory...\n";
    hipFree(devMatrix);
    hipFree(devVector);
    hipFree(devResult);
    free(hostMatrix);
    free(hostVector);
    free(hostResult);

    std::cout << "Done.\n";
    return 0;
}
